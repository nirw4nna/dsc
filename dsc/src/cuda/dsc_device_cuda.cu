#include "hip/hip_runtime.h"
// Copyright (c) 2024-2025, Christian Gilli <christian.gilli11@gmail.com>
// All rights reserved.
//
// This code is licensed under the terms of the 3-clause BSD license
// (https://opensource.org/license/bsd-3-clause).

#include "cuda/dsc_cuda.cuh"
#include "dsc_device.h"

// As per https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#device-memory-accesses
// "Any address of a variable residing in global memory or returned by one of the memory allocation routines
// from the driver or runtime API is always aligned to at least 256 bytes."
#define DSC_DEVICE_CUDA_ALIGN ((usize) 256)
#define DSC_MEMCPY_DIRECTIONS ((int) 4)

static constexpr hipMemcpyKind DSC_CUDA_MEMCPY_DIRECTIONS[DSC_MEMCPY_DIRECTIONS] = {
    hipMemcpyDefault,
    hipMemcpyDeviceToHost,
    hipMemcpyHostToDevice,
    hipMemcpyDeviceToDevice,
};

static DSC_CUDA_KERNEL void k_init_random(hiprandState *state) {
    DSC_CUDA_TID();
    hiprand_init(clock64(), tid, 0, &state[tid]);
}

static void cuda_memcpy(void *dst, const void *src, const usize nb, const dsc_memcpy_dir dir) {
    DSC_CUDA_FAIL_ON_ERROR(hipMemcpy(dst, src, nb, DSC_CUDA_MEMCPY_DIRECTIONS[dir]));
}

static void cuda_memset(void *dst, const int c, const usize nb) {
    DSC_CUDA_FAIL_ON_ERROR(hipMemset(dst, c, nb));
}

static void cuda_dispose(dsc_device *dev) {
    DSC_CUDA_FAIL_ON_ERROR(hipFree(dev->device_mem));

    const dsc_cuda_dev_info *info = (dsc_cuda_dev_info *) dev->extra_info;
    DSC_CUBLAS_FAIL_ON_ERROR(hipblasDestroy(info->cublas_handle));

    DSC_CUDA_FAIL_ON_ERROR(hipFree(info->randState));

    DSC_LOG_INFO("%s:%d device %s disposed",
                 DSC_DEVICE_NAMES[dev->type],
                 info->dev_idx,
                 info->name);
}

dsc_device *dsc_cuda_device(usize mem_size, const int cuda_dev) {
    static dsc_cuda_dev_info extra = {
        .name = {},
        .randState = {},
        .cublas_handle = {},
        .dev_idx = cuda_dev,
    };
    DSC_CUBLAS_FAIL_ON_ERROR(hipblasCreate(&extra.cublas_handle));

    // Allocate 90% of the device memory at most (is this too much?)
    const usize max_mem = (usize) (0.9 * (f64) dsc_cuda_dev_mem(cuda_dev));
    mem_size = mem_size < max_mem ? mem_size : DSC_ALIGN(max_mem - (DSC_DEVICE_CUDA_ALIGN - 1), DSC_DEVICE_CUDA_ALIGN);
    static dsc_device dev = {
        .used_nodes = {},
        .free_nodes = {},
        .head = {},
        .device_mem = {},
        .alignment = DSC_DEVICE_CUDA_ALIGN,
        .extra_info = &extra,
        .mem_size = DSC_ALIGN(mem_size, DSC_DEVICE_CUDA_ALIGN),
        .used_mem = 0,
        .type = CUDA,
        .memcpy = cuda_memcpy,
        .memset = cuda_memset,
        .dispose = cuda_dispose,
    };

    DSC_CUDA_FAIL_ON_ERROR(hipSetDevice(cuda_dev));

    dsc_cuda_dev_name(cuda_dev, extra.name);

    DSC_CUDA_FAIL_ON_ERROR(hipMalloc(&extra.randState, DSC_CUDA_DEFAULT_THREADS * sizeof(hiprandState)));

    k_init_random<<<1, DSC_CUDA_DEFAULT_THREADS>>>(extra.randState);

    dsc_cuda_sync();

    DSC_CUDA_FAIL_ON_ERROR(hipMalloc(&dev.device_mem, dev.mem_size));

    dev.free_nodes[0].size = dev.mem_size;
    dev.free_nodes[0].data = dev.device_mem;
    dev.free_nodes[0].next = nullptr;

    dev.head = &dev.free_nodes[0];

    DSC_LOG_INFO("%s:%d device %s initialized with a buffer of %ldMB (total: %ldMB)",
                 DSC_DEVICE_NAMES[dev.type],
                 cuda_dev,
                 extra.name,
                 (usize) DSC_B_TO_MB(dev.mem_size),
                 (usize) DSC_B_TO_MB(dsc_cuda_dev_mem(cuda_dev)));

    return &dev;
}